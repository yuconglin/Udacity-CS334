#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO  you can clear all the things in this function, which is your homework.
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y; 
  if (idx_x >= numRows || idx_y >= numCols) return;
  int idx_1d = idx_x * numCols + idx_y;
  const uchar4 rgba = rgbaImage[idx_1d];
  greyImage[idx_1d] = rgba.x * .299f + rgba.y * .587f + rgba.z * .114f;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const int block_size = 32;
  int blockrows = ceil(1. * numRows / block_size);
  int blockcols = ceil(1. * numCols / block_size);
  const dim3 blockSize(block_size, block_size, 1);  //TODO
  const dim3 gridSize( blockrows, blockcols, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}

