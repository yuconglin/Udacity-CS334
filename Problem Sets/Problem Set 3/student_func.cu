#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include <limits.h>
#include <float.h>
#include <math.h>
#include <stdio.h>

#include "utils.h"

__global__
void histogram_kernel(unsigned int* d_bins, const float* d_in, const int bin_count, const float lum_min, const float lum_max, const int size) {  
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    float lum_range = lum_max - lum_min;
    int bin = ((d_in[mid]-lum_min) / lum_range) * bin_count;
    
    atomicAdd(&d_bins[bin], 1);
}

__global__ 
void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    
    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_bins[spot];
          __syncthreads();
          if(spot >= 0)
              d_bins[mid] += val;
          __syncthreads();

    }
}
// calculate reduce max or min and stick the value in d_answer.
__global__
void reduce_minmax_kernel(const float* const d_in, float* d_out, const size_t size, int minmax) {
    extern __shared__ float shared[];
    
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x; 
    
    // we have 1 thread per block, so copying the entire block should work fine
    if(mid < size) {
        shared[tid] = d_in[mid];
    } else {
        if(minmax == 0)
            shared[tid] = FLT_MAX;
        else
            shared[tid] = -FLT_MAX;
    }
    
    // wait for all threads to copy the memory
    __syncthreads();
    
    // don't do any thing with memory if we happen to be far off ( I don't know how this works with
    // sync threads so I moved it after that point )
    if(mid >= size) {   
        if(tid == 0) {
            if(minmax == 0) 
                d_out[blockIdx.x] = FLT_MAX;
            else
                d_out[blockIdx.x] = -FLT_MAX;

        }
        return;
    }
       
    for(unsigned int s = blockDim.x/2; s > 0; s /= 2) {
        if(tid < s) {
            if(minmax == 0) {
                shared[tid] = min(shared[tid], shared[tid+s]);
            } else {
                shared[tid] = max(shared[tid], shared[tid+s]);
            }
        }
        
        __syncthreads();
    }
    
    if(tid == 0) {
        d_out[blockIdx.x] = shared[0];
    }
}

int get_max_size(int n, int d) {
    return (int)ceil( (float)n/(float)d ) + 1;
}

float reduce_minmax(const float* const d_in, const size_t size, int minmax) {
    int BLOCK_SIZE = 32;
    // we need to keep reducing until we get to the amount that we consider 
    // having the entire thing fit into one block size
    size_t curr_size = size;
    float* d_curr_in;
    
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice));


    float* d_curr_out;
    
    dim3 thread_dim(BLOCK_SIZE);
    const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
    
    while(1) {
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(curr_size, BLOCK_SIZE)));
        
        dim3 block_dim(get_max_size(size, BLOCK_SIZE));
        reduce_minmax_kernel<<<block_dim, thread_dim, shared_mem_size>>>(
            d_curr_in,
            d_curr_out,
            curr_size,
            minmax
        );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

            
        // move the current input to the output, and clear the last input if necessary
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        if(curr_size <  BLOCK_SIZE) 
            break;
        
        curr_size = get_max_size(curr_size, BLOCK_SIZE);
    }
    
    // theoretically we should be 
    float h_out;
    hipMemcpy(&h_out, d_curr_out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_curr_out);
    return h_out;
}

__global__ void reduce_find_min(const size_t N, float* buf) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    for (int s1 = (N + 1) / 2; s1 > 1; s1 = (s1 + 1) >> 1) {
      if (tid < s1) {
        buf[tid] = min(buf[tid], buf[tid + s1]);
      }
    }
    if (tid == 0)
      buf[0] = min(buf[0], buf[1]);
}

__global__ void reduce_find_max(const size_t N, float* buf) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    for (int s1 = (N + 1) / 2; s1 > 1; s1 = (s1 + 1) >> 1) {
      if (tid < s1) {
        buf[tid] = max(buf[tid], buf[tid + s1]);
      }
    }
    if (tid == 0) 
      buf[0] = max(buf[0], buf[1]);
}


__global__ void reduce_find_max2(const size_t N, const float* const buf, float* d_max) {
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    shared[tid] = -FLT_MAX;

    if (gid < N)
      shared[tid] = buf[gid];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (tid < s && gid < N)
        shared[tid] = max(shared[tid], shared[tid + s]);
      __syncthreads();
    }

    if (tid == 0) 
      d_max[blockIdx.x] = shared[tid];
}

float reduce_max(const float* const d_in, const size_t size, int block_size) {
    // we need to keep reducing until we get to the amount that we consider 
    // having the entire thing fit into one block size
    size_t curr_size = size;
    float* d_curr_in;
    
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice));


    float* d_curr_out;
    
    dim3 thread_dim(block_size);
    const int shared_mem_size = sizeof(float)*block_size;
    
    while(1) {
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(curr_size, block_size)));
        
        dim3 block_dim(get_max_size(size, block_size));
        reduce_find_max2<<<block_dim, thread_dim, shared_mem_size>>>(
	    curr_size,
            d_curr_in,
            d_curr_out
        );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

            
        // move the current input to the output, and clear the last input if necessary
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        if(curr_size < block_size) 
            break;
        
        curr_size = get_max_size(curr_size, block_size);
    }
    
    // theoretically we should be 
    float h_out;
    hipMemcpy(&h_out, d_curr_out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_curr_out);
    return h_out;
}

__global__ void getBinOfInput(const float* const input, float lumMin, float lumRange, size_t numBins, size_t dataCount, unsigned int* col_o) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int bin_num = 0;
    if (tid < dataCount) {
      bin_num = min((size_t)(((input[tid] - lumMin) / lumRange) * numBins), numBins - 1);
      atomicAdd(&col_o[bin_num], 1);
    }
}

__global__ void getCdf(unsigned int *d_bin, const int numBins, unsigned int* const d_cdf) {
    for (int i = 1; i < numBins; ++i) {
      d_cdf[i] = d_bin[i-1] + d_cdf[i-1];
    }
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
    const size_t size = numRows*numCols;
    min_logLum = reduce_minmax(d_logLuminance, size, 0);
    max_logLum = reduce_minmax(d_logLuminance, size, 1);
    
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    printf("numBins %d\n", numBins);
    
    unsigned int* d_bins;
    size_t histo_size = sizeof(unsigned int)*numBins;

    checkCudaErrors(hipMalloc(&d_bins, histo_size));    
    checkCudaErrors(hipMemset(d_bins, 0, histo_size));  
    dim3 thread_dim(1024);
    dim3 hist_block_dim(get_max_size(size, thread_dim.x));
    histogram_kernel<<<hist_block_dim, thread_dim>>>(d_bins, d_logLuminance, numBins, min_logLum, max_logLum, size);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    unsigned int h_out[100];
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    dim3 scan_block_dim(get_max_size(numBins, thread_dim.x));

    scan_kernel<<<scan_block_dim, thread_dim>>>(d_bins, numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);

    hipMemcpy(d_cdf, d_bins, histo_size, hipMemcpyDeviceToDevice);
    
    checkCudaErrors(hipFree(d_bins));
    
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
    //1)
    float *d_buf;
    int N = numRows * numCols;
    if (N & 1) N += 1;
    checkCudaErrors(hipMalloc(&d_buf, sizeof(float) * N));
    hipMemcpy(d_buf, d_logLuminance, numRows * numCols, hipMemcpyDeviceToDevice);
    /*
    // another way
    reduce_find_max<<<(N + 1023) / 1024, 1024>>>(N, d_buf);
    hipMemcpy(&max_logLum, d_buf, sizeof(float), hipMemcpyDeviceToHost);
    */
    max_logLum = reduce_max(d_buf, N, (N+1023)/1024);

    hipMemcpy(d_buf, d_logLuminance, numRows * numCols, hipMemcpyDeviceToDevice);
    reduce_find_min<<<(N + 1023) / 1024, 1024>>>(N, d_buf);
    hipMemcpy(&min_logLum, d_buf, sizeof(float), hipMemcpyDeviceToHost);

    checkCudaErrors(hipFree(d_buf));
    
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    //2
    float lumRange = max_logLum - min_logLum;
    N = numRows * numCols;
    //3
    unsigned int *d_bin;
    checkCudaErrors(hipMalloc(&d_bin, sizeof(unsigned int) * numBins));
    hipMemset(d_bin, 0, sizeof(unsigned int) * numBins);
    getBinOfInput<<<(N + 1023) / 1024, 1024>>>(d_logLuminance, min_logLum, lumRange, numBins, N, d_bin);
    //4
    hipMemset(d_cdf, 0, sizeof(numBins));
    getCdf<<<1, 1>>>(d_bin, numBins, d_cdf);
}
